#include "setGPU.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include <vector>
#include <boost/atomic.hpp>
static boost::atomic<int> activeCudaDeviceIndex;
static std::vector<int> availableGPUs;
bool initGPUSelection()
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        std::cout << "Found "<< deviceProperties.name << " PCI id " << deviceProperties.pciBusID << std::endl;
        if (deviceProperties.major >= 3
            && deviceProperties.minor >= 0)
        {
            std::cout << "Adding "<< deviceProperties.name << " PCI id " << deviceProperties.pciBusID << " to available GPUs" << std::endl;
            availableGPUs.push_back(deviceIndex);
            if(deviceProperties.major < 6){
                std::cout << "Newer GPU architecture recommended (6+)" << std::endl;
            }
        }
    }

    if(availableGPUs.size() == 0){
    std::cerr << "New GPU not found! Exiting." << std::endl;
    exit(1);
    return false;
    }
    else{
        activeCudaDeviceIndex = 0;
        return true;
    }
}

void setGPU(int GPUid){
    hipSetDevice(GPUid);
}

int getGPU(){//int particleNumber, int pCaNum){
    int deviceIndex = availableGPUs[activeCudaDeviceIndex++ % availableGPUs.size()];
    //hipDeviceProp_t deviceProperties;
    //hipGetDeviceProperties(&deviceProperties, deviceIndex);
    //std::cout << "Selecting " << deviceProperties.name << " PCI id " << deviceProperties.pciBusID << " for particle number " << particleNumber << " and cc number " << pCaNum << std::endl;
    return deviceIndex;
}

void printCurrentGPU(){
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, device);
    std::cout << "Current GPU is "<< deviceProperties.name << " PCI id " << deviceProperties.pciBusID << std::endl;
}
