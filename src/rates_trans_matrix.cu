#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------------//
//                    |                                       |                         //
//                    |          Function Name                |                         //
//                    |        rates_trans_matrix ()          |                         //
//                    |                                       |                         //
//--------------------------------------------------------------------------------------//
// Inputs   |
//---------
// parameters / reference values for transition rates
//--------
// Outputs|
//---------
// kB_plus,kB_minus,f,g:  Matrices-->" Tables " of coefficients that depend on Neighboring states (X,Y)
//--------------------------------------------------------------------------
// Notation
//----------
//
//          |-----|-----|-----|-----|-----|-----|
//          | B*  | C*  |  B  |  C  |  M1 |  M2 |
//     |----|-----|-----|-----|-----|-----|-----|
//     | B* |(0,0)|(0,1)|(0,2)|(0,3)|(0,4)|(0,5)|
//     |----|-----|-----|-----|-----|-----|-----|
//     | C* |(1,0)|(1,1)|(1,2)|(1,3)|(1,4)|(1,5)|
//     |----|-----|-----|-----|-----|-----|-----|
//     | B  |(2,0)|(2,1)|(2,2)|(2,3)|(2,4)|(2,5)|
//     |----|-----|-----|-----|-----|-----|-----|
//     | C  |(3,0)|(3,1)|(3,2)|(3,3)|(3,4)|(3,5)|
//     |----|-----|-----|-----|-----|-----|-----|
//     | M1 |(4,0)|(4,1)|(4,2)|(4,3)|(4,4)|(4,5)|
//     |----|-----|-----|-----|-----|-----|-----|
//     | M2 |(4,0)|(4,1)|(4,2)|(4,3)|(4,4)|(5,5)|
//     |----|-----|-----|-----|-----|-----|-----|
//--------------------------------------------------------------------------------
#include "rates_trans_matrix.h"
#include "problemDefines.h"

#include <math.h>
// arr[x*N_S+y] == arr[x * row_len + y]
void rates_trans_matrix(const int n_s,
                        const float kB_plus_ref,
                        const float kB_minus_ref,
                        const float k2_plus_ref_dATP,
                        const float k2_plus_ref_ATP,
                        const float k2_minus_ref,
                        const float k4_plus_ref_dATP,
                        const float k4_plus_ref_ATP,
                        const float k4_minus_ref,
                        const float gamma_B,
                        const float gamma_M,
                        const float mu_B,
                        const float mu_M,
                        const float r,
                        const float q,
                        float *kB_plus,
                        float *kB_minus,
                        float *k2_plus_dATP,
                        float *k2_plus_ATP,
                        float *k2_minus,
                        float *k4_plus_dATP,
                        float *k4_plus_ATP,
                        float *k4_minus
                       )

{
//--------------------------------------------------
// Step 1: Build the kB_plus [ns*N_S+ns] matrix
//--------------------------------------------------
kB_plus[0*N_S+0] = kB_plus_ref*pow(gamma_B,-2*q);
kB_plus[0*N_S+1] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[0*N_S+2] = kB_plus_ref*pow(gamma_B,-2*q);
kB_plus[0*N_S+3] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[0*N_S+4] = kB_plus_ref*pow((gamma_M/gamma_B),q);
kB_plus[0*N_S+5] = kB_plus_ref*pow((gamma_M/gamma_B),q);
//-------
kB_plus[1*N_S+0] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[1*N_S+1] = kB_plus_ref;
kB_plus[1*N_S+2] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[1*N_S+3] = kB_plus_ref;
kB_plus[1*N_S+4] = kB_plus_ref*pow(gamma_M,q);
kB_plus[1*N_S+5] = kB_plus_ref*pow(gamma_M,q);
//-------
kB_plus[2*N_S+0] = kB_plus_ref*pow(gamma_B,-2*q);
kB_plus[2*N_S+1] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[2*N_S+2] = kB_plus_ref*pow(gamma_B,-2*q);
kB_plus[2*N_S+3] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[2*N_S+4] = kB_plus_ref*pow((gamma_M/gamma_B),q);
kB_plus[2*N_S+5] = kB_plus_ref*pow((gamma_M/gamma_B),q);
//-------
kB_plus[3*N_S+0] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[3*N_S+1] = kB_plus_ref;
kB_plus[3*N_S+2] = kB_plus_ref*pow(gamma_B,-q);
kB_plus[3*N_S+3] = kB_plus_ref;
kB_plus[3*N_S+4] = kB_plus_ref*pow(gamma_M,q);
kB_plus[3*N_S+5] = kB_plus_ref*pow(gamma_M,q);
//-------
kB_plus[4*N_S+0] = kB_plus_ref*pow((gamma_M/gamma_B),q);
kB_plus[4*N_S+1] = kB_plus_ref*pow(gamma_M,q);
kB_plus[4*N_S+2] = kB_plus_ref*pow((gamma_M/gamma_B),q);
kB_plus[4*N_S+3] = kB_plus_ref*pow(gamma_M,q);
kB_plus[4*N_S+4] = kB_plus_ref*pow(gamma_M,2*q);
kB_plus[4*N_S+5] = kB_plus_ref*pow(gamma_M,2*q);
//-------
kB_plus[5*N_S+0] = kB_plus_ref*pow((gamma_M/gamma_B),q);
kB_plus[5*N_S+1] = kB_plus_ref*pow(gamma_M,q);
kB_plus[5*N_S+2] = kB_plus_ref*pow((gamma_M/gamma_B),q);
kB_plus[5*N_S+3] = kB_plus_ref*pow(gamma_M,q);
kB_plus[5*N_S+4] = kB_plus_ref*pow(gamma_M,2*q);
kB_plus[5*N_S+5] = kB_plus_ref*pow(gamma_M,2*q);
//---------------------------------------------------

//---------------------------------------------------
// Step 2: Build the kB_minus [ns*N_S+ns] matrix
//---------------------------------------------------
kB_minus[0*N_S+0] = kB_minus_ref*pow(pow(gamma_B,-2),q-1);
kB_minus[0*N_S+1] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[0*N_S+2] = kB_minus_ref*pow(pow(gamma_B,-2),q-1);
kB_minus[0*N_S+3] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[0*N_S+4] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
kB_minus[0*N_S+5] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
//-------
kB_minus[1*N_S+0] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[1*N_S+1] = kB_minus_ref;
kB_minus[1*N_S+2] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[1*N_S+3] = kB_minus_ref;
kB_minus[1*N_S+4] = kB_minus_ref*pow(gamma_M,q-1);
kB_minus[1*N_S+5] = kB_minus_ref*pow(gamma_M,q-1);
//-------
kB_minus[2*N_S+0] = kB_minus_ref*pow(pow(gamma_B,-2),q-1);
kB_minus[2*N_S+1] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[2*N_S+2] = kB_minus_ref*pow(pow(gamma_B,-2),q-1);
kB_minus[2*N_S+3] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[2*N_S+4] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
kB_minus[2*N_S+5] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
//-------
kB_minus[3*N_S+0] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[3*N_S+1] = kB_minus_ref;
kB_minus[3*N_S+2] = kB_minus_ref*pow(1/gamma_B,q-1);
kB_minus[3*N_S+3] = kB_minus_ref;
kB_minus[3*N_S+4] = kB_minus_ref*pow(gamma_M,q-1);
kB_minus[3*N_S+5] = kB_minus_ref*pow(gamma_M,q-1);
//-------
kB_minus[4*N_S+0] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
kB_minus[4*N_S+1] = kB_minus_ref*pow(gamma_M,q-1);
kB_minus[4*N_S+2] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
kB_minus[4*N_S+3] = kB_minus_ref*pow(gamma_M,q-1);
kB_minus[4*N_S+4] = kB_minus_ref*pow(pow(gamma_M,2),q-1);
kB_minus[4*N_S+5] = kB_minus_ref*pow(pow(gamma_M,2),q-1);
//-------
kB_minus[5*N_S+0] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
kB_minus[5*N_S+1] = kB_minus_ref*pow(gamma_M,q-1);
kB_minus[5*N_S+2] = kB_minus_ref*pow(gamma_M/gamma_B,q-1);
kB_minus[5*N_S+3] = kB_minus_ref*pow(gamma_M,q-1);
kB_minus[5*N_S+4] = kB_minus_ref*pow(pow(gamma_M,2),q-1);
kB_minus[5*N_S+5] = kB_minus_ref*pow(pow(gamma_M,2),q-1);
//-----------------------------------

//--------------------------------------------------
// Step 3: Build the k2_plus [ns*N_S+ns] matrix
//--------------------------------------------------
k2_plus_ATP[0*N_S+0] = k2_plus_ref_ATP*pow(mu_B,-2*r);
k2_plus_ATP[0*N_S+1] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[0*N_S+2] = k2_plus_ref_ATP*pow(mu_B,-2*r);
k2_plus_ATP[0*N_S+3] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[0*N_S+4] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
k2_plus_ATP[0*N_S+5] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
//-------
k2_plus_ATP[1*N_S+0] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[1*N_S+1] = k2_plus_ref_ATP;
k2_plus_ATP[1*N_S+2] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[1*N_S+3] = k2_plus_ref_ATP;
k2_plus_ATP[1*N_S+4] = k2_plus_ref_ATP*pow(mu_M,r);
k2_plus_ATP[1*N_S+5] = k2_plus_ref_ATP*pow(mu_M,r);
//-------
k2_plus_ATP[2*N_S+0] = k2_plus_ref_ATP*pow(mu_B,-2*r);
k2_plus_ATP[2*N_S+1] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[2*N_S+2] = k2_plus_ref_ATP*pow(mu_B,-2*r);
k2_plus_ATP[2*N_S+3] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[2*N_S+4] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
k2_plus_ATP[2*N_S+5] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
//-------
k2_plus_ATP[3*N_S+0] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[3*N_S+1] = k2_plus_ref_ATP;
k2_plus_ATP[3*N_S+2] = k2_plus_ref_ATP*pow(mu_B,-r);
k2_plus_ATP[3*N_S+3] = k2_plus_ref_ATP;
k2_plus_ATP[3*N_S+4] = k2_plus_ref_ATP*pow(mu_M,r);
k2_plus_ATP[3*N_S+5] = k2_plus_ref_ATP*pow(mu_M,r);
//-------
k2_plus_ATP[4*N_S+0] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
k2_plus_ATP[4*N_S+1] = k2_plus_ref_ATP*pow(mu_M,r);
k2_plus_ATP[4*N_S+2] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
k2_plus_ATP[4*N_S+3] = k2_plus_ref_ATP*pow(mu_M,r);
k2_plus_ATP[4*N_S+4] = k2_plus_ref_ATP*pow(mu_M,2*r);
k2_plus_ATP[4*N_S+5] = k2_plus_ref_ATP*pow(mu_M,2*r);
//-------
k2_plus_ATP[5*N_S+0] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
k2_plus_ATP[5*N_S+1] = k2_plus_ref_ATP*pow(mu_M,r);
k2_plus_ATP[5*N_S+2] = k2_plus_ref_ATP*pow((mu_M/mu_B),r);
k2_plus_ATP[5*N_S+3] = k2_plus_ref_ATP*pow(mu_M,r);
k2_plus_ATP[5*N_S+4] = k2_plus_ref_ATP*pow(mu_M,2*r);
k2_plus_ATP[5*N_S+5] = k2_plus_ref_ATP*pow(mu_M,2*r);
//-------



k2_plus_dATP[0*N_S+0] = k2_plus_ref_dATP*pow(mu_B,-2*r);
k2_plus_dATP[0*N_S+1] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[0*N_S+2] = k2_plus_ref_dATP*pow(mu_B,-2*r);
k2_plus_dATP[0*N_S+3] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[0*N_S+4] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
k2_plus_dATP[0*N_S+5] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
//-------
k2_plus_dATP[1*N_S+0] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[1*N_S+1] = k2_plus_ref_dATP;
k2_plus_dATP[1*N_S+2] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[1*N_S+3] = k2_plus_ref_dATP;
k2_plus_dATP[1*N_S+4] = k2_plus_ref_dATP*pow(mu_M,r);
k2_plus_dATP[1*N_S+5] = k2_plus_ref_dATP*pow(mu_M,r);
//-------
k2_plus_dATP[2*N_S+0] = k2_plus_ref_dATP*pow(mu_B,-2*r);
k2_plus_dATP[2*N_S+1] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[2*N_S+2] = k2_plus_ref_dATP*pow(mu_B,-2*r);
k2_plus_dATP[2*N_S+3] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[2*N_S+4] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
k2_plus_dATP[2*N_S+5] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
//-------
k2_plus_dATP[3*N_S+0] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[3*N_S+1] = k2_plus_ref_dATP;
k2_plus_dATP[3*N_S+2] = k2_plus_ref_dATP*pow(mu_B,-r);
k2_plus_dATP[3*N_S+3] = k2_plus_ref_dATP;
k2_plus_dATP[3*N_S+4] = k2_plus_ref_dATP*pow(mu_M,r);
k2_plus_dATP[3*N_S+5] = k2_plus_ref_dATP*pow(mu_M,r);
//-------
k2_plus_dATP[4*N_S+0] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
k2_plus_dATP[4*N_S+1] = k2_plus_ref_dATP*pow(mu_M,r);
k2_plus_dATP[4*N_S+2] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
k2_plus_dATP[4*N_S+3] = k2_plus_ref_dATP*pow(mu_M,r);
k2_plus_dATP[4*N_S+4] = k2_plus_ref_dATP*pow(mu_M,2*r);
k2_plus_dATP[4*N_S+5] = k2_plus_ref_dATP*pow(mu_M,2*r);
//-------
k2_plus_dATP[5*N_S+0] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
k2_plus_dATP[5*N_S+1] = k2_plus_ref_dATP*pow(mu_M,r);
k2_plus_dATP[5*N_S+2] = k2_plus_ref_dATP*pow((mu_M/mu_B),r);
k2_plus_dATP[5*N_S+3] = k2_plus_ref_dATP*pow(mu_M,r);
k2_plus_dATP[5*N_S+4] = k2_plus_ref_dATP*pow(mu_M,2*r);
k2_plus_dATP[5*N_S+5] = k2_plus_ref_dATP*pow(mu_M,2*r);
//---------------------------------------------------

//---------------------------------------------------
// Step 4: Build the k2_minus [ns*N_S+ns] matrix
//---------------------------------------------------
k2_minus[0*N_S+0] = k2_minus_ref*pow(pow(mu_B,-2),r-1);
k2_minus[0*N_S+1] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[0*N_S+2] = k2_minus_ref*pow(pow(mu_B,-2),r-1);
k2_minus[0*N_S+3] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[0*N_S+4] = k2_minus_ref*pow(mu_M/mu_B,r-1);
k2_minus[0*N_S+5] = k2_minus_ref*pow(mu_M/mu_B,r-1);
//-------
k2_minus[1*N_S+0] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[1*N_S+1] = k2_minus_ref;
k2_minus[1*N_S+2] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[1*N_S+3] = k2_minus_ref;
k2_minus[1*N_S+4] = k2_minus_ref*pow(mu_M,r-1);
k2_minus[1*N_S+5] = k2_minus_ref*pow(mu_M,r-1);
//-------
k2_minus[2*N_S+0] = k2_minus_ref*pow(pow(mu_B,-2),r-1);
k2_minus[2*N_S+1] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[2*N_S+2] = k2_minus_ref*pow(pow(mu_B,-2),r-1);
k2_minus[2*N_S+3] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[2*N_S+4] = k2_minus_ref*pow(mu_M/mu_B,r-1);
k2_minus[2*N_S+5] = k2_minus_ref*pow(mu_M/mu_B,r-1);
//-------
k2_minus[3*N_S+0] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[3*N_S+1] = k2_minus_ref;
k2_minus[3*N_S+2] = k2_minus_ref*pow(1/mu_B,r-1);
k2_minus[3*N_S+3] = k2_minus_ref;
k2_minus[3*N_S+4] = k2_minus_ref*pow(mu_M,r-1);
k2_minus[3*N_S+5] = k2_minus_ref*pow(mu_M,r-1);
//-------
k2_minus[4*N_S+0] = k2_minus_ref*pow(mu_M/mu_B,r-1);
k2_minus[4*N_S+1] = k2_minus_ref*pow(mu_M,r-1);
k2_minus[4*N_S+2] = k2_minus_ref*pow(mu_M/mu_B,r-1);
k2_minus[4*N_S+3] = k2_minus_ref*pow(mu_M,r-1);
k2_minus[4*N_S+4] = k2_minus_ref*pow(pow(mu_M,2),r-1);
k2_minus[4*N_S+5] = k2_minus_ref*pow(pow(mu_M,2),r-1);
//-------
k2_minus[5*N_S+0] = k2_minus_ref*pow(mu_M/mu_B,r-1);
k2_minus[5*N_S+1] = k2_minus_ref*pow(mu_M,r-1);
k2_minus[5*N_S+2] = k2_minus_ref*pow(mu_M/mu_B,r-1);
k2_minus[5*N_S+3] = k2_minus_ref*pow(mu_M,r-1);
k2_minus[5*N_S+4] = k2_minus_ref*pow(pow(mu_M,2),r-1);
k2_minus[5*N_S+5] = k2_minus_ref*pow(pow(mu_M,2),r-1);
//-----------------------------------

//---------------------------------------------------
// Step 5: Build the k4_plus [ns*N_S+ns] matrix
//---------------------------------------------------
k4_plus_ATP[0*N_S+0] = k4_plus_ref_ATP*pow(pow(mu_B,-2),-.5);
k4_plus_ATP[0*N_S+1] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[0*N_S+2] = k4_plus_ref_ATP*pow(pow(mu_B,-2),-.5);
k4_plus_ATP[0*N_S+3] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[0*N_S+4] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
k4_plus_ATP[0*N_S+5] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
//-------
k4_plus_ATP[1*N_S+0] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[1*N_S+1] = k4_plus_ref_ATP;
k4_plus_ATP[1*N_S+2] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[1*N_S+3] = k4_plus_ref_ATP;
k4_plus_ATP[1*N_S+4] = k4_plus_ref_ATP*pow(mu_M,-.5);
k4_plus_ATP[1*N_S+5] = k4_plus_ref_ATP*pow(mu_M,-.5);
//-------
k4_plus_ATP[2*N_S+0] = k4_plus_ref_ATP*pow(pow(mu_B,-2),-.5);
k4_plus_ATP[2*N_S+1] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[2*N_S+2] = k4_plus_ref_ATP*pow(pow(mu_B,-2),-.5);
k4_plus_ATP[2*N_S+3] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[2*N_S+4] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
k4_plus_ATP[2*N_S+5] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
//-------
k4_plus_ATP[3*N_S+0] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[3*N_S+1] = k4_plus_ref_ATP;
k4_plus_ATP[3*N_S+2] = k4_plus_ref_ATP*pow(1/mu_B,-.5);
k4_plus_ATP[3*N_S+3] = k4_plus_ref_ATP;
k4_plus_ATP[3*N_S+4] = k4_plus_ref_ATP*pow(mu_M,-.5);
k4_plus_ATP[3*N_S+5] = k4_plus_ref_ATP*pow(mu_M,-.5);
//-------
k4_plus_ATP[4*N_S+0] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
k4_plus_ATP[4*N_S+1] = k4_plus_ref_ATP*pow(mu_M,-.5);
k4_plus_ATP[4*N_S+2] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
k4_plus_ATP[4*N_S+3] = k4_plus_ref_ATP*pow(mu_M,-.5);
k4_plus_ATP[4*N_S+4] = k4_plus_ref_ATP*pow(pow(mu_M,2),-.5);
k4_plus_ATP[4*N_S+5] = k4_plus_ref_ATP*pow(pow(mu_M,2),-.5);
//-------
k4_plus_ATP[5*N_S+0] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
k4_plus_ATP[5*N_S+1] = k4_plus_ref_ATP*pow(mu_M,-.5);
k4_plus_ATP[5*N_S+2] = k4_plus_ref_ATP*pow(mu_M/mu_B,-.5);
k4_plus_ATP[5*N_S+3] = k4_plus_ref_ATP*pow(mu_M,-.5);
k4_plus_ATP[5*N_S+4] = k4_plus_ref_ATP*pow(pow(mu_M,2),-.5);
k4_plus_ATP[5*N_S+5] = k4_plus_ref_ATP*pow(pow(mu_M,2),-.5);
//-------




k4_plus_dATP[0*N_S+0] = k4_plus_ref_dATP*pow(pow(mu_B,-2),-.5);
k4_plus_dATP[0*N_S+1] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[0*N_S+2] = k4_plus_ref_dATP*pow(pow(mu_B,-2),-.5);
k4_plus_dATP[0*N_S+3] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[0*N_S+4] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
k4_plus_dATP[0*N_S+5] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
//-------
k4_plus_dATP[1*N_S+0] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[1*N_S+1] = k4_plus_ref_dATP;
k4_plus_dATP[1*N_S+2] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[1*N_S+3] = k4_plus_ref_dATP;
k4_plus_dATP[1*N_S+4] = k4_plus_ref_dATP*pow(mu_M,-.5);
k4_plus_dATP[1*N_S+5] = k4_plus_ref_dATP*pow(mu_M,-.5);
//-------
k4_plus_dATP[2*N_S+0] = k4_plus_ref_dATP*pow(pow(mu_B,-2),-.5);
k4_plus_dATP[2*N_S+1] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[2*N_S+2] = k4_plus_ref_dATP*pow(pow(mu_B,-2),-.5);
k4_plus_dATP[2*N_S+3] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[2*N_S+4] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
k4_plus_dATP[2*N_S+5] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
//-------
k4_plus_dATP[3*N_S+0] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[3*N_S+1] = k4_plus_ref_dATP;
k4_plus_dATP[3*N_S+2] = k4_plus_ref_dATP*pow(1/mu_B,-.5);
k4_plus_dATP[3*N_S+3] = k4_plus_ref_dATP;
k4_plus_dATP[3*N_S+4] = k4_plus_ref_dATP*pow(mu_M,-.5);
k4_plus_dATP[3*N_S+5] = k4_plus_ref_dATP*pow(mu_M,-.5);
//-------
k4_plus_dATP[4*N_S+0] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
k4_plus_dATP[4*N_S+1] = k4_plus_ref_dATP*pow(mu_M,-.5);
k4_plus_dATP[4*N_S+2] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
k4_plus_dATP[4*N_S+3] = k4_plus_ref_dATP*pow(mu_M,-.5);
k4_plus_dATP[4*N_S+4] = k4_plus_ref_dATP*pow(pow(mu_M,2),-.5);
k4_plus_dATP[4*N_S+5] = k4_plus_ref_dATP*pow(pow(mu_M,2),-.5);
//-------
k4_plus_dATP[5*N_S+0] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
k4_plus_dATP[5*N_S+1] = k4_plus_ref_dATP*pow(mu_M,-.5);
k4_plus_dATP[5*N_S+2] = k4_plus_ref_dATP*pow(mu_M/mu_B,-.5);
k4_plus_dATP[5*N_S+3] = k4_plus_ref_dATP*pow(mu_M,-.5);
k4_plus_dATP[5*N_S+4] = k4_plus_ref_dATP*pow(pow(mu_M,2),-.5);
k4_plus_dATP[5*N_S+5] = k4_plus_ref_dATP*pow(pow(mu_M,2),-.5);
//-----------------------------------

//--------------------------------------------------
// Step 6: Build the k4_minus [ns*N_S+ns] matrix
//--------------------------------------------------
k4_minus[0*N_S+0] = k4_minus_ref*pow(mu_B,-2);
k4_minus[0*N_S+1] = k4_minus_ref*pow(mu_B,-1);
k4_minus[0*N_S+2] = k4_minus_ref*pow(mu_B,-2);
k4_minus[0*N_S+3] = k4_minus_ref*pow(mu_B,-1);
k4_minus[0*N_S+4] = k4_minus_ref*pow((mu_M/mu_B),1);
k4_minus[0*N_S+5] = k4_minus_ref*pow((mu_M/mu_B),1);
//-------
k4_minus[1*N_S+0] = k4_minus_ref*pow(mu_B,-1);
k4_minus[1*N_S+1] = k4_minus_ref;
k4_minus[1*N_S+2] = k4_minus_ref*pow(mu_B,-1);
k4_minus[1*N_S+3] = k4_minus_ref;
k4_minus[1*N_S+4] = k4_minus_ref*pow(mu_M,1);
k4_minus[1*N_S+5] = k4_minus_ref*pow(mu_M,1);
//-------
k4_minus[2*N_S+0] = k4_minus_ref*pow(mu_B,-2);
k4_minus[2*N_S+1] = k4_minus_ref*pow(mu_B,-1);
k4_minus[2*N_S+2] = k4_minus_ref*pow(mu_B,-2);
k4_minus[2*N_S+3] = k4_minus_ref*pow(mu_B,-1);
k4_minus[2*N_S+4] = k4_minus_ref*pow((mu_M/mu_B),1);
k4_minus[2*N_S+5] = k4_minus_ref*pow((mu_M/mu_B),1);
//-------
k4_minus[3*N_S+0] = k4_minus_ref*pow(mu_B,-1);
k4_minus[3*N_S+1] = k4_minus_ref;
k4_minus[3*N_S+2] = k4_minus_ref*pow(mu_B,-1);
k4_minus[3*N_S+3] = k4_minus_ref;
k4_minus[3*N_S+4] = k4_minus_ref*pow(mu_M,1);
k4_minus[3*N_S+5] = k4_minus_ref*pow(mu_M,1);
//-------
k4_minus[4*N_S+0] = k4_minus_ref*pow((mu_M/mu_B),1);
k4_minus[4*N_S+1] = k4_minus_ref*pow(mu_M,1);
k4_minus[4*N_S+2] = k4_minus_ref*pow((mu_M/mu_B),1);
k4_minus[4*N_S+3] = k4_minus_ref*pow(mu_M,1);
k4_minus[4*N_S+4] = k4_minus_ref*pow(mu_M,2);
k4_minus[4*N_S+5] = k4_minus_ref*pow(mu_M,2);
//-------
k4_minus[5*N_S+0] = k4_minus_ref*pow((mu_M/mu_B),1);
k4_minus[5*N_S+1] = k4_minus_ref*pow(mu_M,1);
k4_minus[5*N_S+2] = k4_minus_ref*pow((mu_M/mu_B),1);
k4_minus[5*N_S+3] = k4_minus_ref*pow(mu_M,1);
k4_minus[5*N_S+4] = k4_minus_ref*pow(mu_M,2);
k4_minus[5*N_S+5] = k4_minus_ref*pow(mu_M,2);
//---------------------------------------------------

}
