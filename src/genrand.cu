#include "genrand.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
__device__ void genrand(float * numbers, const size_t numbersLen, hiprandState * state)
{
    for(int i = 0; i < numbersLen; ++i)
    {
        numbers[i] = hiprand_uniform(state);
    }
}
